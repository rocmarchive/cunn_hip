
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/AbsCriterion.cu"
#else

void THNN_(AbsCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 2, input, target);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

#if THRUST_PATH
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), abs_functor<real, accreal>());
#else
  auto input_data = bolt::amp::make_ubiquitous_iterator(THCTensor_(data)(state, input));
  auto target_data = bolt::amp::make_ubiquitous_iterator(THCTensor_(data)(state, target));
  accreal sum = bolt::amp::inner_product(input_data, input_data+size, target_data, (accreal)0, bolt::amp::plus<accreal>(), abs_functor<real, accreal>());
#endif

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(AbsCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  real norm = ScalarConvert<double, real>::to(sizeAverage ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

#if THRUST_PATH
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor<real>(norm));
#else
  auto input_data = bolt::amp::make_ubiquitous_iterator(THCTensor_(data)(state, input));
  auto target_data = bolt::amp::make_ubiquitous_iterator(THCTensor_(data)(state, target));
  auto gradInput_data = bolt::amp::make_ubiquitous_iterator(THCTensor_(data)(state, gradInput));

  bolt::amp::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor<real>(norm));
#endif

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
